#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 8192
#define BATCH 128


extern "C" void* runCudaPart(void* _src);
extern "C" void cudaInit();
extern "C" void cudaExit();

float *pfHost;
float *pfDevice;
float *pfSrcDevice;
int nBlocks,nThreads,nSize,n;

__global__ void my_kernel(float*x , void* src)
{
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    x[tid] = *((float*)src + tid);
}

void cudaInit()
{
    gpuDeviceInit(0);
    nBlocks = 2;
    nThreads = 8;
    nSize = nBlocks * nThreads;
    hipHostAlloc(&pfHost,nSize*sizeof(float),hipHostMallocDefault);
    hipMalloc((void**)&pfDevice,nSize*sizeof(float));
    hipMalloc((void**)&pfSrcDevice,nSize*sizeof(float));
    cout<<"cudaInit"<<endl;
}

void cudaExit()
{
    hipFree(pfHost);
    hipFree(pfDevice);
    hipFree(pfSrcDevice);
    cout<<"cudaExit"<<endl;
}

void* runCudaPart(void *_src)
{
    hipMemcpy(pfSrcDevice,_src,nSize * sizeof(float),hipMemcpyHostToDevice);
    my_kernel <<< nBlocks, nThreads >>> (pfDevice,pfSrcDevice);
    hipMemcpy(pfHost,pfDevice,nSize * sizeof(float),hipMemcpyDeviceToHost);

    hipfftHandle plan;
    hipfftComplex *data,*pHostData;
    hipMalloc((void**)&data,sizeof(hipfftComplex)*NX*BATCH);
    hipHostAlloc(&pHostData,sizeof(hipfftComplex)*NX*BATCH,hipHostMallocDefault);

    for(int i=0;i<NX;i++)
    {
        pHostData[i].x  = 1000*((float *)_src)[i];
        pHostData[i].y = 0;
    }

    if ( hipGetLastError() != hipSuccess )
        cout<<"Cuda e r r o r : F a i l e d t o a l l o c a t e "<<endl  ;

    if ( hipfftPlan1d(&plan , NX , HIPFFT_C2C , BATCH ) != HIPFFT_SUCCESS )
        cout<<"CUFFT e r r o r : Plan c r e a t i o n f a i l e d "<<endl  ;

    hipMemcpy(data,pHostData,sizeof(hipfftComplex)*NX*BATCH,hipMemcpyHostToDevice);
    if ( hipfftExecC2C ( plan , data , data , HIPFFT_FORWARD ) != HIPFFT_SUCCESS )
        cout<<"CUFFT e r r o r : ExecC2C Forward f a i l e d "<<endl ;

    if ( hipfftExecC2C ( plan , data , data , HIPFFT_BACKWARD ) != HIPFFT_SUCCESS )
        cout<<"CUFFT e r r o r : ExecC2C I n v e r s e f a i l e d "<<endl  ;

    if ( hipDeviceSynchronize() != hipSuccess )
        cout<<"Cuda e r r o r : F a i l e d t o s y n c h r o n i z e " <<endl ;

    hipMemcpy(pHostData,data,sizeof(hipfftComplex)*NX*BATCH,hipMemcpyDeviceToHost);

    for(int i=0;i<NX;i++)
    {
        //pfHost[i] = sqrt(pHostData[i].x*pHostData[i].x + pHostData[i].y*pHostData[i].y);
        cout<<i<<' '<<pHostData[i].x <<' '<<pHostData[i].y<<' '<<pfHost[i]<<endl;
        pfHost[i]=pHostData[i].y;
    }

    hipfftDestroy ( plan ) ;
    hipFree ( data ) ;


    cout<<"runCuda"<<endl;
    return pfHost;
}
